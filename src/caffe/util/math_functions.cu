#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

//----------- MY max -----------------

template <>
void caffe_gpu_max<float>(const int n, const float* x, int* y) {
  CUBLAS_CHECK(hipblasIsamax(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_max<double>(const int n, const double* x, int* y) {
  CUBLAS_CHECK(hipblasIdamax(Caffe::cublas_handle(), n, x, 1, y));
}
//--------- end my max-----------------------------


//----------- MY min -----------------

template <>
void caffe_gpu_min<float>(const int n, const float* x, int* y) {
  CUBLAS_CHECK(hipblasIsamin(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_min<double>(const int n, const double* x, int* y) {
  CUBLAS_CHECK(hipblasIdamin(Caffe::cublas_handle(), n, x, 1, y));
}
//--------- end my min-----------------------------


// ------------ My add --------------------------
template <typename Dtype>
__global__ void my_add_kernel(const int n, const Dtype* a,
    Dtype b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b;
  }
}

template <>
void my_caffe_gpu_add<float>(const int N, const float* a,
    float b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  my_add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void my_caffe_gpu_add<double>(const int N, const double* a,
    double b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  my_add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}
// --------------- END div ------------------------------



// ------------ My div --------------------------
template <typename Dtype>
__global__ void my_div_kernel(const int n, const Dtype* a,
    Dtype b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b;
  }
}

template <>
void my_caffe_gpu_div<float>(const int N, const float* a,
    float b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  my_div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void my_caffe_gpu_div<double>(const int N, const double* a,
    double b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  my_div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}
// --------------- END div ------------------------------


// ------------ My Reg --------------------------
template <typename Dtype>
__global__ void my_reg_kernel_pos(const int n, const Dtype* a,
    Dtype max, Dtype min, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = ((a[index] - min) / (max - min));
  }
}


template <typename Dtype>
__global__ void my_reg_kernel_neg(const int n, const Dtype* a,
    Dtype max, Dtype min, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = ((a[index] - min) / (max - min))*(0 - (-1)) + (-1);
  }
}


template <typename Dtype>
__global__ void my_reg_kernel_all(const int n, const Dtype* a,
    Dtype max, Dtype min, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = ((a[index] - min) / (max - min))*(1 - (-1)) + (-1);
  }
}

template <>
void my_caffe_gpu_reg<float>(const int N, const float* a,
    float b, float c, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
	if (c>=0 && b >=0)
	{
		my_reg_kernel_pos<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}
	else if (c<0 && b>=0){
		my_reg_kernel_all<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}
	else if (c<0 && b<0){
		my_reg_kernel_neg<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}

}

template <>
void my_caffe_gpu_reg<double>(const int N, const double* a,
    double b, double c, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
	if (c>=0 && b >=0)
	{
		my_reg_kernel_pos<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}
	else if (c<0 && b>=0){
		my_reg_kernel_all<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}
	else if (c<0 && b<0){
		my_reg_kernel_neg<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}
	// Old one
	/*
	if (c>=0)
	{
		my_reg_kernel_pos<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}
	else{
		my_reg_kernel_all<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, c, y);
	}
	*/
}
// --------------- END Reg ------------------------------


// ------------ My New Min --------------------------
template <typename Dtype>
__global__ void my_new_min_kernel(const int n, const Dtype* a,
    Dtype *max, Dtype *min, Dtype* y) {
	Dtype min_=10000;
	Dtype max_=-1000;
	Dtype sum=0.0;
  for (int i=0;i<n;i++) {
	  if (a[i]<min_){
//		  min_ = a[i];
	  }
	  if (a[i]>max_){
//		  max_=a[i];
	  }
	  sum=sum+a[i];
  }
  *max=max_;
  *min=min_;
}

template <>
void my_caffe_gpu_new_min<float>(const int N, const float* a,
    float *b, float *c, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
//	float aa= *a;
	thrust::device_ptr<float> min_ptr = thrust::min_element(thrust::device_pointer_cast(y), thrust::device_pointer_cast(y) + N);
	thrust::device_ptr<float> max_ptr = thrust::max_element(thrust::device_pointer_cast(y), thrust::device_pointer_cast(y) + N);
	*c=min_ptr[0];
	*b = max_ptr[0];
//	*b=2;
//  my_new_min_kernel<float><<<CAFFE_GET_BLOCKS(N), 0>>>(    N, a, b, c, y);
}

template <>
void my_caffe_gpu_new_min<double>(const int N, const double* a,
    double *b, double *c, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
//	double aa=*a;
	thrust::device_ptr<double> min_ptr = thrust::min_element(thrust::device_pointer_cast(y), thrust::device_pointer_cast(y) + N);
	thrust::device_ptr<double> max_ptr = thrust::max_element(thrust::device_pointer_cast(y), thrust::device_pointer_cast(y) + N);
	*c=min_ptr[0];
	*b = max_ptr[0];
//	*b=2;
//  	my_new_min_kernel<double><<<CAFFE_GET_BLOCKS(N), 0>>>( N, a, b, c, y);
}
// --------------- END New Min ------------------------------


}  // namespace caffe
